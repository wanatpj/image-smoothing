#include "hip/hip_runtime.h"
extern "C" {

__device__ double sobely_kernel[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};
__device__ double sobelx_kernel[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
__device__ double sharpen_kernel[3][3] = {{0, -1, 0}, {-1, 5, -1}, {0, -1, 0}};
__device__ double blur_kernel[3][3] =
    {{0, 0.2, 0}, {0.2, 0.2, 0.2}, {0, 0.2, 0}};

__device__
void simple_convolve(const double* channel,
    int dimx,
    const double kernel[3][3],
    int y,
    int x,
    double * out) {
  double result = 0;
  for (int i = -1; i <= 1; ++i) {
    for (int j = -1; j <= 1; ++j) {
      result += channel[(y + i) * dimx + x + j] * kernel[i + 1][j + 1];
    }
  }
  *out = result;
}

__global__
void sobel_filter(const double* in_channel,
    int dimy,
    int dimx,
    double* out_channel) {
  int idy = blockIdx.y*32 + threadIdx.y;
  int idx = blockIdx.x*32 + threadIdx.x;
  if (idy < dimy - 1 && idx < dimx - 1 && 0 < idy && 0 < idx) {
    double sobely;
    double sobelx;
    simple_convolve(in_channel,
        dimx,
        sobely_kernel,
        idy,
        idx,
        &sobely);
    simple_convolve(in_channel,
        dimx,
        sobelx_kernel,
        idy,
        idx,
        &sobelx);
    out_channel[idy * dimx + idx] = sqrtf(sobely*sobely + sobelx*sobelx);
  } else if (idy < dimy && idx < dimx) {
    out_channel[idy * dimx + idx] = 0;
  }
}

__global__
void blur_filter(const double* in_channel,
    int dimy,
    int dimx,
    double* out_channel) {
  int idy = blockIdx.y*32 + threadIdx.y;
  int idx = blockIdx.x*32 + threadIdx.x;
  if (idy < dimy - 1 && idx < dimx - 1 && 0 < idy && 0 < idx) {
    double result;
    simple_convolve(in_channel,
        dimx,
        blur_kernel,
        idy,
        idx,
        &result);
    out_channel[idy * dimx + idx] = result;
  } else if (idy < dimy && idx < dimx) {
    out_channel[idy * dimx + idx] = in_channel[idy * dimx + idx];
  }
}

__global__
void mix_channels(const double * rchannels,
    const double * gchannels,
    const double * bchannels,
    int len,
    double* out_channel) {
  int id = blockIdx.x*1024 + threadIdx.x;
  if (id < len) {
    out_channel[id] = rchannels[id] * 0.33
        + gchannels[id] * 0.34
        + bchannels[id] * 0.33;
  }
}

__global__
void smoothen(const double* in_channel,
    int dimy,
    int dimx,
    const double* edge_channel,
    unsigned char* out_channel) {
  int idy = blockIdx.y*32 + threadIdx.y;
  int idx = blockIdx.x*32 + threadIdx.x;
  if (idy >= dimy || idx >= dimx) {
    return;
  }
  double result = 0;
  if (idy < dimy - 1 && idx < dimx - 1 && 0 < idy && 0 < idx) {
    simple_convolve(in_channel,
        dimx,
        (edge_channel[idy * dimx + idx] >= 30 ? sharpen_kernel : blur_kernel),
        idy,
        idx,
        &result);
  } else {
    result = in_channel[idy * dimx + idx];
  }
  out_channel[idy * dimx + idx] =
      (unsigned char)max(0., min(255., round(result)));
}

__global__
void parallel_max(const double* in, int length, double* out) {
  __shared__ double values[1024];
  int i = 1;
  int idx = blockIdx.x*1024 + threadIdx.x;
  values[threadIdx.x] = idx < length ? in[idx] : 0;
  __syncthreads();
  while (i < 1024) {
    if (threadIdx.x + i < 1024) {
      values[threadIdx.x] = max(values[threadIdx.x], values[threadIdx.x + i]);
    }
    i *= 2;
    __syncthreads();
  }
  out[blockIdx.x] = values[0];
}

__global__
void divide_all(double * array, int length, double* divider) {  // remove *
  int idx = blockIdx.x*1024 + threadIdx.x;
  if (idx < length) {
    array[idx] *= 255./(*divider);
  }
}
}
